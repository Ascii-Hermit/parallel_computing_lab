#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>

__global__ void reduceWord(char* word, char* reducedWord, int wordLength) {
    int idx = threadIdx.x; 

    if (idx < wordLength) {
        for (int i = 0; i < wordLength - idx; i++) {
            reducedWord[idx * wordLength + i] = word[i];
        }
    }
}

int main() {
    char word[] = "AMBUJ SHUKLA";
    int wordLength = strlen(word);
    
    char *d_word, *d_reducedWord;
    hipMalloc((void**)&d_word, wordLength * sizeof(char));
    hipMalloc((void**)&d_reducedWord, wordLength * wordLength * sizeof(char));

    hipMemcpy(d_word, word, wordLength * sizeof(char), hipMemcpyHostToDevice);

    reduceWord<<<1, wordLength>>>(d_word, d_reducedWord, wordLength);

    char reducedWord[wordLength * wordLength];
    
    hipMemcpy(reducedWord, d_reducedWord, wordLength * wordLength * sizeof(char), hipMemcpyDeviceToHost);

    printf("Reduced word steps:\n");
    for (int i = 0; i < wordLength; i++) {
        for (int j = 0; j < wordLength - i; j++) {
            printf("%c", reducedWord[i * wordLength + j]);
        }
        if(i != wordLength-1)
            printf("->");
    }
    printf("\n");

    hipFree(d_word);
    hipFree(d_reducedWord);

    return 0;
}
