#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void sort(int *arr, int l_arr, int *ans) {
    int i = (blockDim.x * blockIdx.x) + threadIdx.x;

    if (i < l_arr) {
        int val = arr[i];
        int idx = 0;

        for (int j = 0; j < l_arr; ++j) {
            if (arr[j] < val || (arr[j] == val && j < i)) {
                idx += 1;
            }
        }

        ans[idx] = val;
    }
}

int main() {
    int *arr, *ans, n;

    printf("Enter size of array:\n");
    scanf("%d", &n);

    arr = (int *)malloc(sizeof(int) * n);
    ans = (int *)malloc(sizeof(int) * n);

    printf("Enter array elements:\n");
    for (int i = 0; i < n; ++i) {
        scanf("%d", &arr[i]);
        ans[i] = -1; 
    }

    int *darr, *dans;
    hipMalloc((void **)&darr, n * sizeof(int));
    hipMalloc((void **)&dans, n * sizeof(int));

    hipMemcpy(darr, arr, n * sizeof(int), hipMemcpyHostToDevice);

    sort<<<1, n>>>(darr, n, dans);
    hipDeviceSynchronize();

    hipMemcpy(ans, dans, n * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(darr);
    hipFree(dans);

    printf("Sorted output:\n");
    for (int i = 0; i < n; ++i) {
        printf("%d ", ans[i]);
    }
    printf("\n");

    free(arr);
    free(ans);

    return 0;
}
