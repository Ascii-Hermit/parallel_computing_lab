hipEvent_t start, stop;
hipEventCreate(&start);
hipEventCreate(&stop);

hipEventRecord(start); // Record start time

dim3 gridDim(ROWS);
dim3 blockDim(COLS);
matrixMultiply<<<gridDim, blockDim>>>(d_A, d_B, d_C);

hipEventRecord(stop);  // Record stop time
hipDeviceSynchronize(); // Wait for the kernel to finish

// Calculate elapsed time
float milliseconds = 0;
hipEventElapsedTime(&milliseconds, start, stop);
printf("Time taken for matrix multiplication: %f ms\n", milliseconds);