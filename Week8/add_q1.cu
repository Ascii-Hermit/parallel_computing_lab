#include <stdio.h>
#include <hip/hip_runtime.h>

#define ROWS 3
#define COLS 3

//row sums kernel
__global__ void computeRowSums(int *matrix, int *rowSums, int rows, int cols) {
    int row = threadIdx.x;
    if (row < rows) {
        int sum = 0;
        for (int j = 0; j < cols; j++) {
            sum += matrix[row * cols + j];
        }
        rowSums[row] = sum;
    }
}

//column sums kernel
__global__ void computeColumnSums(int *matrix, int *colSums, int rows, int cols) {
    int col = threadIdx.x;
    if (col < cols) {
        int sum = 0;
        for (int i = 0; i < rows; i++) {
            sum += matrix[i * cols + col];
        }
        colSums[col] = sum;
    }
}

__global__ void replaceElements(int *matrix, int *rowSums, int *colSums, int rows, int cols) {
    int row = blockIdx.x;
    int col = threadIdx.x;

    if (row < rows && col < cols) {
        int index = row * cols + col;
        if (matrix[index] % 2 == 0) {
            matrix[index] = rowSums[row];  // replace row sum
        } else {
            matrix[index] = colSums[col];  // replace column sum
        }
    }
}

void printMatrix(int *matrix, int rows, int cols) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            printf("%d ", matrix[i * cols + j]);
        }
        printf("\n");
    }
}

int main() {
    int matrix[ROWS][COLS] = {{1, 2, 3}, {4, 5, 6}, {7, 8, 9}};
    int rowSums[ROWS], colSums[COLS];

    int *d_matrix, *d_rowSums, *d_colSums;
    size_t sizeMatrix = ROWS * COLS * sizeof(int);
    size_t sizeRowSums = ROWS * sizeof(int);
    size_t sizeColSums = COLS * sizeof(int);

    hipMalloc((void **)&d_matrix, sizeMatrix);
    hipMalloc((void **)&d_rowSums, sizeRowSums);
    hipMalloc((void **)&d_colSums, sizeColSums);

    hipMemcpy(d_matrix, matrix, sizeMatrix, hipMemcpyHostToDevice);

    printf("Original Matrix:\n");
    printMatrix((int *)matrix, ROWS, COLS);

    computeRowSums<<<1, ROWS>>>(d_matrix, d_rowSums, ROWS, COLS);
    hipMemcpy(rowSums, d_rowSums, sizeRowSums, hipMemcpyDeviceToHost);

    computeColumnSums<<<1, COLS>>>(d_matrix, d_colSums, ROWS, COLS);
    hipMemcpy(colSums, d_colSums, sizeColSums, hipMemcpyDeviceToHost);

    dim3 gridDim(ROWS);
    dim3 blockDim(COLS);
    replaceElements<<<gridDim, blockDim>>>(d_matrix, d_rowSums, d_colSums, ROWS, COLS);

    hipMemcpy(matrix, d_matrix, sizeMatrix, hipMemcpyDeviceToHost);

    printf("\nModified Matrix:\n");
    printMatrix((int *)matrix, ROWS, COLS);

    hipFree(d_matrix);
    hipFree(d_rowSums);
    hipFree(d_colSums);

    return 0;
}
