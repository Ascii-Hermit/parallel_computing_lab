#include <stdio.h>
#include <hip/hip_runtime.h>

#define ROWS 3
#define COLS 3

// i)Each thread computes one row
__global__ void addRowWise(int *A, int *B, int *C, int rows, int cols) {
    int row = threadIdx.x;
    if (row < rows) {
        for (int j = 0; j < cols; j++) {
            C[row * cols + j] = A[row * cols + j] + B[row * cols + j];
        }
    }
}

// ii)Each thread computes one column
__global__ void addColumnWise(int *A, int *B, int *C, int rows, int cols) {
    int col = threadIdx.x;
    if (col < cols) {
        for (int i = 0; i < rows; i++) {
            C[i * cols + col] = A[i * cols + col] + B[i * cols + col];
        }
    }
}

// iii)Each thread computes one element
__global__ void addElementWise(int *A, int *B, int *C, int rows, int cols) {
    int row = blockIdx.x;
    int col = threadIdx.x;
    if (row < rows && col < cols) {
        C[row * cols + col] = A[row * cols + col] + B[row * cols + col];
    }
}

void printMatrix(int *matrix, int rows, int cols) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            printf("%d ", matrix[i * cols + j]);
        }
        printf("\n");
    }
}

int main() {
    int A[ROWS][COLS] = {{1, 2, 3}, {4, 5, 6}, {7, 8, 9}};
    int B[ROWS][COLS] = {{9, 8, 7}, {6, 5, 4}, {3, 2, 1}};
    int C[ROWS][COLS];

    int *d_A, *d_B, *d_C;
    size_t size = ROWS * COLS * sizeof(int);

    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    printf("Matrix A:\n");
    printMatrix((int *)A, ROWS, COLS);
    printf("\nMatrix B:\n");
    printMatrix((int *)B, ROWS, COLS);

    addRowWise<<<1, ROWS>>>(d_A, d_B, d_C, ROWS, COLS);
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
    printf("\nRow-wise Addition:\n");
    printMatrix((int *)C, ROWS, COLS);

    addColumnWise<<<1, COLS>>>(d_A, d_B, d_C, ROWS, COLS);
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
    printf("\nColumn-wise Addition:\n");
    printMatrix((int *)C, ROWS, COLS);

    dim3 gridDim(ROWS);
    dim3 blockDim(COLS); // not necesary to do dim3, but easier to understand
    addElementWise<<<gridDim, blockDim>>>(d_A, d_B, d_C, ROWS, COLS);
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
    printf("\nElement-wise Addition:\n");
    printMatrix((int *)C, ROWS, COLS);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
