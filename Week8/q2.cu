#include <stdio.h>
#include <hip/hip_runtime.h>

#define ROWS 3
#define COLS 3
#define COMMON_DIM 3 //A: ROWS × COMMON_DIM, B: COMMON_DIM × COLS

// i)Each thread computes one row of the result matrix
__global__ void multiplyRowWise(int *A, int *B, int *C, int rows, int cols, int common) {
    int row = threadIdx.x;
    if (row < rows) {
        for (int j = 0; j < cols; j++) {
            int sum = 0;
            for (int k = 0; k < common; k++) {
                sum += A[row * common + k] * B[k * cols + j];
            }
            C[row * cols + j] = sum;
        }
    }
}

// ii)Each thread computes one column of the result matrix
__global__ void multiplyColumnWise(int *A, int *B, int *C, int rows, int cols, int common) {
    int col = threadIdx.x;
    if (col < cols) {
        for (int i = 0; i < rows; i++) {
            int sum = 0;
            for (int k = 0; k < common; k++) {
                sum += A[i * common + k] * B[k * cols + col];
            }
            C[i * cols + col] = sum;
        }
    }
}

// iii) Each thread computes one element of the result matrix
__global__ void multiplyElementWise(int *A, int *B, int *C, int rows, int cols, int common) {
    int row = blockIdx.x;
    int col = threadIdx.x;
    if (row < rows && col < cols) {
        int sum = 0;
        for (int k = 0; k < common; k++) {
            sum += A[row * common + k] * B[k * cols + col];
        }
        C[row * cols + col] = sum;
    }
}

void printMatrix(int *matrix, int rows, int cols) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            printf("%d ", matrix[i * cols + j]);
        }
        printf("\n");
    }
}

int main() {
    int A[ROWS][COMMON_DIM] = {{1, 2, 3}, {4, 5, 6}, {7, 8, 9}};
    int B[COMMON_DIM][COLS] = {{9, 8, 7}, {6, 5, 4}, {3, 2, 1}};
    int C[ROWS][COLS];

    int *d_A, *d_B, *d_C;
    size_t sizeA = ROWS * COMMON_DIM * sizeof(int);
    size_t sizeB = COMMON_DIM * COLS * sizeof(int);
    size_t sizeC = ROWS * COLS * sizeof(int);

    hipMalloc((void **)&d_A, sizeA);
    hipMalloc((void **)&d_B, sizeB);
    hipMalloc((void **)&d_C, sizeC);

    hipMemcpy(d_A, A, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, sizeB, hipMemcpyHostToDevice);

    printf("Matrix A:\n");
    printMatrix((int *)A, ROWS, COMMON_DIM);
    printf("\nMatrix B:\n");
    printMatrix((int *)B, COMMON_DIM, COLS);

    multiplyRowWise<<<1, ROWS>>>(d_A, d_B, d_C, ROWS, COLS, COMMON_DIM);
    hipMemcpy(C, d_C, sizeC, hipMemcpyDeviceToHost);
    printf("\nRow-wise Multiplication:\n");
    printMatrix((int *)C, ROWS, COLS);

    multiplyColumnWise<<<1, COLS>>>(d_A, d_B, d_C, ROWS, COLS, COMMON_DIM);
    hipMemcpy(C, d_C, sizeC, hipMemcpyDeviceToHost);
    printf("\nColumn-wise Multiplication:\n");
    printMatrix((int *)C, ROWS, COLS);

    dim3 gridDim(ROWS);
    dim3 blockDim(COLS);
    multiplyElementWise<<<gridDim, blockDim>>>(d_A, d_B, d_C, ROWS, COLS, COMMON_DIM);
    hipMemcpy(C, d_C, sizeC, hipMemcpyDeviceToHost);
    printf("\nElement-wise Multiplication:\n");
    printMatrix((int *)C, ROWS, COLS);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
