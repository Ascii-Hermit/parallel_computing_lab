#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int *a, int *b, int *c, int N) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        c[idx] = a[idx] + b[idx];
    }
    //printf("hello\n");
}

int main() {
    int N = 1000;
    int size = N * sizeof(int);

    int *h_a, *h_b, *h_c;
    int *d_a, *d_b, *d_c;

    // Allocate memory on the host
    h_a = (int *)malloc(size);
    h_b = (int *)malloc(size);
    h_c = (int *)malloc(size);

    // Initialize host arrays
    for (int i = 0; i < N; i++) {
        h_a[i] = i;
        h_b[i] = i * 2;
    }

    // Allocate memory on the device
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // Copy data from host to device
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    // Launch kernel with 256 threads per block and enough blocks to cover N elements
    add<<<(N + 255) / 256, 256>>>(d_a, d_b, d_c, N);

    // Copy the result from device to host
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    // Print the first few results
    for (int i = 0; i < 10; i++) {
        printf("c[%d] = %d\n", i, h_c[i]);
    }

    // Free memory
    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
