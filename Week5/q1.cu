#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void vector_addition(int *A, int *B, int *C, int N) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) { 
        C[idx] = A[idx] + B[idx];
    }
}

int main() {
    int N = 1000; 
    int size = N * sizeof(int); 
    int *h_A, *h_B, *h_C;
    h_A = (int*)malloc(size);
    h_B = (int*)malloc(size);
    h_C = (int*)malloc(size);

    for (int i = 0; i < N; i++) {
        h_A[i] = i;      
        h_B[i] = i * 3;  
    }

    int *d_A, *d_B, *d_C;

    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;  
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;  

    vector_addition<<<N, threadsPerBlock>>>(d_A, d_B, d_C, N);
    vector_addition<<<blocksPerGrid, N>>>(d_A, d_B, d_C, N);

    hipDeviceSynchronize();

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    printf("Result:\n");
    for (int i = 0; i < 10; i++) {
        printf("C[%d] = %d\n", i, h_C[i]);
    }

    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
